#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstdio>
#include <cmath>
#include <fstream>
#include <iostream>
#include <time.h>
#include <hipDNN.h>
#include <string>
#include <vector>
#include <algorithm>
#include "json/json.h"
using namespace std;
 
#define ErrChk(code)                                                           \
  { Assert((code), __FILE__, __LINE__); }
inline void Assert(hipError_t code, const char *file, int line) {
  if (code != hipSuccess) {
    printf("CUDA Runtime Error: %s:%d:'%s'\n", file, line,
           hipGetErrorString(code));
    // exit(EXIT_FAILURE);
  }
}
inline void Assert(hipdnnStatus_t code, const char *file, int line) {
  if (code != HIPDNN_STATUS_SUCCESS) {
    printf("cuDNN API Error: %s:%d:'%s'\n", file, line,
           hipdnnGetErrorString(code));
    // exit(EXIT_FAILURE);
  }
}

inline int _ConvertSMVer2Cores(int major, int minor) { //根据GPU Arch确定每个SM上有多少个SP
  // Defines for GPU Architecture types (using the SM version to determine
  // the # of cores per SM
  typedef struct {
    int SM; // 0xMm (hexidecimal notation), M = SM Major version,
    // and m = SM minor version
    int Cores;
  } sSMtoCores;

  sSMtoCores nGpuArchCoresPerSM[] = { { 0x30, 192 },
                                      { 0x32, 192 },
                                      { 0x35, 192 },
                                      { 0x37, 192 },
                                      { 0x50, 128 },
                                      { 0x52, 128 },
                                      { 0x53, 128 },
                                      { 0x60, 64 },
                                      { 0x61, 128 },
                                      { 0x62, 128 },
                                      { 0x70, 64 },
                                      { 0x72, 64 },
                                      { 0x75, 64 },
                                      { -1, -1 } };

  int index = 0;

  while (nGpuArchCoresPerSM[index].SM != -1) {
    if (nGpuArchCoresPerSM[index].SM == ((major << 4) + minor)) {
      return nGpuArchCoresPerSM[index].Cores;
    }

    index++;
  }

  // If we don't find the values, we default use the previous one
  // to run properly
  printf("MapSMtoCores for SM %d.%d is undefined."
         "  Default to use %d Cores/SM\n",
         major, minor, nGpuArchCoresPerSM[index - 1].Cores);
  return nGpuArchCoresPerSM[index - 1].Cores; //如果没找到 就选择0x75对应的64
}

int main(int argc, char **argv) {

  printf("%s Starting...\n\n", argv[0]);
  printf("CUDA Device Query (Runtime API) version (CUDART static linking)\n\n");

  int deviceCount = 0;
  ErrChk(hipGetDeviceCount(&deviceCount));
  if (deviceCount == 0) {
    printf("There are no available device(s) that support CUDA\n");
  } else {
    printf("Detected %d CUDA Capable device(s)\n", deviceCount);
  }

  int dev = 0;
  for (dev = 0; dev < deviceCount; ++dev) {

    ErrChk(hipSetDevice(dev));

    hipDeviceProp_t deviceProp;
    ErrChk(hipGetDeviceProperties(&deviceProp, dev));

    printf("\nDevice %d: \"%s\"\n", dev, deviceProp.name);
    printf("GPU Max Clock rate: %0.2f (GHz)\n", deviceProp.clockRate * 1e-6f);

    float freq = deviceProp.clockRate * 1e-6f; // GHz
    int nCore = _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor);
    int nSM = deviceProp.multiProcessorCount;
    printf("(%2d) Multiprocessors, (%2d) CUDA Cores/MP: %d CUDA Cores\n", nSM,
           nCore, nSM * nCore);

    float peakPerf = nSM * nCore * freq * 2;

    printf("GPU Peak Performance is %0.2f GFlops.\n", peakPerf);
  }

  // cuDNN implementation

  // ErrChk(hipSetDevice(dev));

  hipdnnTensorFormat_t format = HIPDNN_TENSOR_NCHW;

  Json::Reader reader;
  Json::FastWriter fwriter;
  Json::Value op_samples;
  Json::Value result_json;
  
  std::ifstream i("./conv2d_all.json");
  if (!reader.parse(i, op_samples, false))
  {
      printf("parse failed! \n");
  }
  i.close();

  std::ofstream result_csv("./conv2d_results_2080ti.csv");
  result_csv << "Batch_sizeN,input_channel,image_size_H,image_size_W,output_channel,kernel_sizeR,kernel_sizeS,strideU,strideV,pad_h,pad_w,algo0,algo1,algo2,algo3,algo4,algo5,algo6,algo7,algoSelected,algoMin\n";
  int BATCH = op_samples["input_size"].size();
  for (int sample_iter = 0; sample_iter < BATCH; sample_iter++)
  {
    int N = op_samples["input_size"][sample_iter][0].asInt();
    int C = op_samples["input_size"][sample_iter][1].asInt();//输入通道
    int H = op_samples["input_size"][sample_iter][2].asInt();
    int W = op_samples["input_size"][sample_iter][3].asInt();

    int K = op_samples["kernel_size"][sample_iter][0].asInt();//输出通道
    int R = op_samples["kernel_size"][sample_iter][2].asInt();//filter长
    int S = op_samples["kernel_size"][sample_iter][3].asInt();//filter宽

    int U = op_samples["stride"][sample_iter][0].asInt();//stride
    int V = op_samples["stride"][sample_iter][1].asInt();//stride
    int pad_h = op_samples["padding"][sample_iter][0].asInt();
    int pad_w = op_samples["padding"][sample_iter][1].asInt();

    int P = floor((H + 2 * pad_h - R) / U) + 1;//输出
    int Q = floor((W + 2 * pad_w - S) / V) + 1;//输出

    std::string config = "["+std::to_string(N) +", "+std::to_string(C)+", "+std::to_string(H)+", "+std::to_string(W)+"]"+" / kernel: ["+std::to_string(K)+", "+std::to_string(R)+", "+std::to_string(S)+"] / stride:"+to_string(U)+", pad, "+std::to_string(pad_h)+"\n";
    std::string config_csv = std::to_string(N) +","+std::to_string(C)+","+std::to_string(H)+","+std::to_string(W)+","+std::to_string(K)+","+std::to_string(R)+","+std::to_string(S)+","+to_string(U)+","+to_string(V)+","+std::to_string(pad_h)+","+std::to_string(pad_w)+",";
    result_csv << config_csv;

    int kernel_size = K * C * R * S;
    int input_size = N * C * H * W ;
    int output_size = N * K * P * Q;


    // alloc host memory
    float *h_Var0 = (float *)malloc(sizeof(float) * input_size);
    if (h_Var0 == NULL) {
      printf("Error in alloc h_Var0\n");
      exit(EXIT_FAILURE);
    }
    float *h_Var1 = (float *)malloc(sizeof(float) * kernel_size);
    if (h_Var1 == NULL) {
      printf("Error in alloc h_Var1\n");
      exit(EXIT_FAILURE);
    }
    float *h_Var2 = (float *)malloc(sizeof(float) * output_size);
    if (h_Var2 == NULL) {
      printf("Error in alloc h_Var2\n");
      exit(EXIT_FAILURE);
    }

    // generate dummy data for test
    for (int i = 0; i < input_size; ++i) {
      h_Var0[i] = static_cast<float>(rand()) / static_cast<float>(RAND_MAX);
      // h_Var0[i] = static_cast <float> (1);
    }
    for (int i = 0; i < kernel_size; ++i) {
      h_Var1[i] = static_cast<float>(rand()) / static_cast<float>(RAND_MAX);
      // h_Var1[i] = static_cast <float> (1);
    }
    for (int i = 0; i < output_size; ++i) {
      h_Var2[i] = static_cast<float>(0);
    }
    float *y, *filter, *x;

    ErrChk(hipMalloc((void **)&y, sizeof(float) * output_size));
    ErrChk(hipMalloc((void **)&filter, sizeof(float) * kernel_size));
    ErrChk(hipMalloc((void **)&x, sizeof(float) * input_size));

    ErrChk(hipMemcpy(x, h_Var0, sizeof(float) * input_size, hipMemcpyHostToDevice));
    ErrChk(hipMemcpy(filter, h_Var1, sizeof(float) * kernel_size, hipMemcpyHostToDevice));
    ErrChk(hipMemcpy(y, h_Var2, sizeof(float) * output_size, hipMemcpyHostToDevice));

    /*  2. cuDNN preparation  */
    hipdnnHandle_t handle;
    ErrChk(hipdnnCreate(&handle));

    float one = 1.0, zero = 0.0;

    hipdnnTensorDescriptor_t yDesc;
    ErrChk(hipdnnCreateTensorDescriptor(&yDesc));
    ErrChk(
        hipdnnSetTensor4dDescriptor(yDesc, format, HIPDNN_DATA_FLOAT, N, K, P, Q));

    hipdnnFilterDescriptor_t filterDesc;
    ErrChk(hipdnnCreateFilterDescriptor(&filterDesc));
    ErrChk(hipdnnSetFilter4dDescriptor(filterDesc, HIPDNN_DATA_FLOAT,
                                      HIPDNN_TENSOR_NCHW, K, C, R, S));
    hipdnnTensorDescriptor_t xDesc;
    ErrChk(hipdnnCreateTensorDescriptor(&xDesc));
    ErrChk(hipdnnSetTensor4dDescriptor(xDesc, format, HIPDNN_DATA_FLOAT, N, C, H, W));

    hipdnnConvolutionDescriptor_t convDesc;
    ErrChk(hipdnnCreateConvolutionDescriptor(&convDesc));
    ErrChk(hipdnnSetConvolution2dDescriptor(convDesc, pad_h, pad_w, U, V, 1, 1,
                                          HIPDNN_CROSS_CORRELATION,
                                          HIPDNN_DATA_FLOAT));
    ErrChk(hipdnnSetConvolutionMathType(convDesc, CUDNN_FMA_MATH)); // turn off tensor core

    const int n_requestedAlgo = 10;
    int n_returnedAlgo;
    hipdnnConvolutionFwdAlgoPerf_t fwd_algo_perf[n_requestedAlgo];
    ErrChk(hipdnnFindConvolutionForwardAlgorithm(
        handle, xDesc, filterDesc, convDesc, yDesc, n_requestedAlgo,
        &n_returnedAlgo, fwd_algo_perf));
    printf("algo choice: %d, time: %f\n", fwd_algo_perf[0].algo, fwd_algo_perf[0].time);
    if (n_returnedAlgo == 0) {
      printf("No hipdnnConvolutionFwdAlgoPerf_t found");
      exit(1);
    }
    float *extra;

    // hipdnnConvolutionFwdAlgo_t fwd_algo = fwd_algo_perf[0].algo;
    std::vector<float> result_vec(8, MAXFLOAT);
    for (int algo_i = 0; algo_i < 8; algo_i++)
    {
      try
      {
        hipdnnConvolutionFwdAlgo_t fwd_algo = hipdnnConvolutionFwdAlgo_t(algo_i);
        printf("algo choice: %d\t", fwd_algo);
        size_t fwd_workspace_size;
        hipdnnStatus_t retStatus = hipdnnGetConvolutionForwardWorkspaceSize(
            handle, xDesc, filterDesc, convDesc, yDesc, fwd_algo,
            &fwd_workspace_size);
        printf("worksapce: %.2fMB\t", (float)fwd_workspace_size/1024/1024);
        if (retStatus != HIPDNN_STATUS_SUCCESS || (float)fwd_workspace_size/1024/1024 > 30000)
        {
          printf("\n");
          continue;
        }
        ErrChk(hipMalloc((void **)&extra, fwd_workspace_size));

        float time_sum = 0.0;
        int count = 0;
        for (int i = 0; i < 10; i++)
        {
          hipEvent_t start, stop;
          hipEventCreate(&start);
          hipEventCreate(&stop) ;
          hipEventRecord(start, 0) ;
          ErrChk(hipdnnConvolutionForward(handle, &one, xDesc, x, filterDesc, filter, convDesc, fwd_algo, extra, fwd_workspace_size, &zero, yDesc, y));
          hipEventRecord(stop, 0) ;
          hipEventSynchronize(stop);
          float elapsedTime;
          hipEventElapsedTime(&elapsedTime, start, stop);
          if (i >= 1)
          {
            count++;
            time_sum += elapsedTime;
          }
          // printf("algo time cost: %f\n", double(end - start)/CLOCKS_PER_SEC);
        }
        
        result_vec[algo_i] = double(time_sum)/count;
        printf("algo time avg cost: %lf ms\n", double(time_sum)/count);
        
        hipFree(extra);
      }
      catch(const char *msg)
      {
        printf("%s", msg);
      }
      
    }
    for (int i = 0; i < 8; i++)
    {
      if (result_vec[i] == MAXFLOAT)
        result_csv << "NOT SUPPORT!,";
      else
        result_csv << std::to_string(result_vec[i])+",";
    }
    
    int minElementIndex = std::min_element(result_vec.begin(), result_vec.end()) - result_vec.begin();
    result_json[std::to_string(minElementIndex)] = result_json[std::to_string(minElementIndex)].asString()+config;
    result_csv << std::to_string(fwd_algo_perf[0].algo)+",";
    result_csv << std::to_string(minElementIndex)+",";
    result_csv << "\n";


    hipFree(y);
    hipFree(filter);
    hipFree(x);
    free(h_Var0);
    free(h_Var1);
    free(h_Var2);
    ErrChk(hipdnnDestroyTensorDescriptor(xDesc));
    ErrChk(hipdnnDestroyTensorDescriptor(yDesc));
    ErrChk(hipdnnDestroyFilterDescriptor(filterDesc));
    ErrChk(hipdnnDestroyConvolutionDescriptor(convDesc));
    printf("%d\n", sample_iter);
  }

  std::ofstream result_txt;
  for (int out_i = 0; out_i < 8; out_i++)
  {
    result_txt.open("./output/best_algo/algo_"+std::to_string(out_i)+".txt", ios::out);
    result_txt << result_json[std::to_string(out_i)].asString();
    result_txt.close();
  }
  result_csv.close();
  exit(EXIT_SUCCESS);
}