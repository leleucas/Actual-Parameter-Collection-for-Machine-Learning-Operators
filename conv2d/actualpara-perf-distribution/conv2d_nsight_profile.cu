#include "hip/hip_runtime.h"
#include <cstdlib>
#include <cstdio>
#include <cmath>
#include <fstream>
#include <iostream>
#include <time.h>
#include <hipDNN.h>
#include <string>
#include <vector>
#include <algorithm>
#include "/home/liuhangda/jsoncpp/jsoncpp-master/include/json/json.h"
using namespace std;

#define ErrChk(code)                                                           \
  { Assert((code), __FILE__, __LINE__); }
inline void Assert(hipError_t code, const char *file, int line) {
  if (code != hipSuccess) {
    printf("CUDA Runtime Error: %s:%d:'%s'\n", file, line,
           hipGetErrorString(code));
    // exit(EXIT_FAILURE);
  }
}
inline void Assert(hipdnnStatus_t code, const char *file, int line) {
  if (code != HIPDNN_STATUS_SUCCESS) {
    printf("cuDNN API Error: %s:%d:'%s'\n", file, line,
           hipdnnGetErrorString(code));
    // exit(EXIT_FAILURE);
  }
}

inline int _ConvertSMVer2Cores(int major, int minor) { //根据GPU Arch确定每个SM上有多少个SP
  // Defines for GPU Architecture types (using the SM version to determine
  // the # of cores per SM
  typedef struct {
    int SM; // 0xMm (hexidecimal notation), M = SM Major version,
    // and m = SM minor version
    int Cores;
  } sSMtoCores;

  sSMtoCores nGpuArchCoresPerSM[] = { { 0x30, 192 },
                                      { 0x32, 192 },
                                      { 0x35, 192 },
                                      { 0x37, 192 },
                                      { 0x50, 128 },
                                      { 0x52, 128 },
                                      { 0x53, 128 },
                                      { 0x60, 64 },
                                      { 0x61, 128 },
                                      { 0x62, 128 },
                                      { 0x70, 64 },
                                      { 0x72, 64 },
                                      { 0x75, 64 },
                                      { -1, -1 } };

  int index = 0;

  while (nGpuArchCoresPerSM[index].SM != -1) {
    if (nGpuArchCoresPerSM[index].SM == ((major << 4) + minor)) {
      return nGpuArchCoresPerSM[index].Cores;
    }

    index++;
  }

  // If we don't find the values, we default use the previous one
  // to run properly
  printf("MapSMtoCores for SM %d.%d is undefined."
         "  Default to use %d Cores/SM\n",
         major, minor, nGpuArchCoresPerSM[index - 1].Cores);
  return nGpuArchCoresPerSM[index - 1].Cores; //如果没找到 就选择0x75对应的64
}

int main(int argc, char **argv) {

  printf("%s Starting...\n\n", argv[0]);
  printf("CUDA Device Query (Runtime API) version (CUDART static linking)\n\n");

  int deviceCount = 0;
  ErrChk(hipGetDeviceCount(&deviceCount));
  if (deviceCount == 0) {
    printf("There are no available device(s) that support CUDA\n");
  } else {
    printf("Detected %d CUDA Capable device(s)\n", deviceCount);
  }

  int dev = 0;
  for (dev = 0; dev < 1; ++dev) {

    ErrChk(hipSetDevice(dev));

    hipDeviceProp_t deviceProp;
    ErrChk(hipGetDeviceProperties(&deviceProp, dev));

    printf("\nDevice %d: \"%s\"\n", dev, deviceProp.name);
    printf("GPU Max Clock rate: %0.2f (GHz)\n", deviceProp.clockRate * 1e-6f);

    float freq = deviceProp.clockRate * 1e-6f; // GHz
    int nCore = _ConvertSMVer2Cores(deviceProp.major, deviceProp.minor);
    int nSM = deviceProp.multiProcessorCount;
    printf("(%2d) Multiprocessors, (%2d) CUDA Cores/MP: %d CUDA Cores\n", nSM,
           nCore, nSM * nCore);

    float peakPerf = nSM * nCore * freq * 2;

    printf("GPU Peak Performance is %0.2f GFlops.\n", peakPerf);
  }

  // cuDNN implementation

  // ErrChk(hipSetDevice(dev));

  hipdnnTensorFormat_t format = HIPDNN_TENSOR_NCHW;

  Json::Reader reader;
  Json::FastWriter fwriter;
  Json::Value op_samples;
  Json::Value result_json;
  
  std::ifstream i("./top3000.json");
  if (!reader.parse(i, op_samples, false))
  {
      printf("parse failed! \n");
  }
  i.close();
  clock_t start, end;
  start = clock();
  int BATCH = op_samples["N"].size();
  for (int sample_iter = 0; sample_iter < BATCH; sample_iter++)
  {
    int N = op_samples["N"][sample_iter][0].asInt();
    int C = op_samples["C_in"][sample_iter][0].asInt();//输入通道
    int H = op_samples["H"][sample_iter][0].asInt();
    int W = op_samples["W"][sample_iter][0].asInt();

    int K = op_samples["C_out"][sample_iter][0].asInt();//输出通道
    int R = op_samples["kernel_R"][sample_iter][0].asInt();//filter长
    int S = op_samples["kernel_S"][sample_iter][0].asInt();//filter宽

    int U = op_samples["strideU"][sample_iter][0].asInt();//stride
    int V = op_samples["strideV"][sample_iter][0].asInt();//stride
    int pad_h = op_samples["pad_h"][sample_iter][0].asInt();
    int pad_w = op_samples["pad_w"][sample_iter][0].asInt();
    int algo_t = op_samples["algoMin"][sample_iter][0].asInt();

    int P = floor((H + 2 * pad_h - R) / U) + 1;//输出
    int Q = floor((W + 2 * pad_w - S) / V) + 1;//输出

    int kernel_size = K * C * R * S;
    int input_size = N * C * H * W ;
    int output_size = N * K * P * Q;


    // alloc host memory
    float *h_Var0 = (float *)malloc(sizeof(float) * input_size);
    if (h_Var0 == NULL) {
      printf("Error in alloc h_Var0 %d %d %d %d %d %d %d\n", K, C, R, S, N, H, W);
      exit(EXIT_FAILURE);
    }
    float *h_Var1 = (float *)malloc(sizeof(float) * kernel_size);
    if (h_Var1 == NULL) {
      printf("Error in alloc h_Var1\n");
      exit(EXIT_FAILURE);
    }
    float *h_Var2 = (float *)malloc(sizeof(float) * output_size);
    if (h_Var2 == NULL) {
      printf("Error in alloc h_Var2\n");
      exit(EXIT_FAILURE);
    }

    // generate dummy data for test
    for (int i = 0; i < input_size; ++i) {
      h_Var0[i] = static_cast<float>(rand()) / static_cast<float>(RAND_MAX);
      // h_Var0[i] = static_cast <float> (1);
    }
    for (int i = 0; i < kernel_size; ++i) {
      h_Var1[i] = static_cast<float>(rand()) / static_cast<float>(RAND_MAX);
      // h_Var1[i] = static_cast <float> (1);
    }
    for (int i = 0; i < output_size; ++i) {
      h_Var2[i] = static_cast<float>(0);
    }
    float *y, *filter, *x;

    ErrChk(hipMalloc((void **)&y, sizeof(float) * output_size));
    ErrChk(hipMalloc((void **)&filter, sizeof(float) * kernel_size));
    ErrChk(hipMalloc((void **)&x, sizeof(float) * input_size));

    ErrChk(hipMemcpy(x, h_Var0, sizeof(float) * input_size, hipMemcpyHostToDevice));
    ErrChk(hipMemcpy(filter, h_Var1, sizeof(float) * kernel_size, hipMemcpyHostToDevice));
    ErrChk(hipMemcpy(y, h_Var2, sizeof(float) * output_size, hipMemcpyHostToDevice));

    /*  2. cuDNN preparation  */
    hipdnnHandle_t handle;
    ErrChk(hipdnnCreate(&handle));

    float one = 1.0, zero = 0.0;

    hipdnnTensorDescriptor_t yDesc;
    ErrChk(hipdnnCreateTensorDescriptor(&yDesc));
    ErrChk(
        hipdnnSetTensor4dDescriptor(yDesc, format, HIPDNN_DATA_FLOAT, N, K, P, Q));

    hipdnnFilterDescriptor_t filterDesc;
    ErrChk(hipdnnCreateFilterDescriptor(&filterDesc));
    ErrChk(hipdnnSetFilter4dDescriptor(filterDesc, HIPDNN_DATA_FLOAT,
                                      HIPDNN_TENSOR_NCHW, K, C, R, S));
    hipdnnTensorDescriptor_t xDesc;
    ErrChk(hipdnnCreateTensorDescriptor(&xDesc));
    ErrChk(hipdnnSetTensor4dDescriptor(xDesc, format, HIPDNN_DATA_FLOAT, N, C, H, W));

    hipdnnConvolutionDescriptor_t convDesc;
    ErrChk(hipdnnCreateConvolutionDescriptor(&convDesc));
    ErrChk(hipdnnSetConvolution2dDescriptor(convDesc, pad_h, pad_w, U, V, 1, 1,
                                          HIPDNN_CROSS_CORRELATION,
                                          HIPDNN_DATA_FLOAT));
    ErrChk(hipdnnSetConvolutionMathType(convDesc, CUDNN_FMA_MATH)); // turn off tensor core
    float *extra;

    //hipdnnConvolutionFwdAlgo_t fwd_algo = fwd_algo_perf[0].algo;
    {
      try
      {
        hipdnnConvolutionFwdAlgo_t fwd_algo = hipdnnConvolutionFwdAlgo_t(algo_t);
        //printf("algo choice: %d\t", fwd_algo);
        size_t fwd_workspace_size;
        hipdnnStatus_t retStatus = hipdnnGetConvolutionForwardWorkspaceSize(
            handle, xDesc, filterDesc, convDesc, yDesc, fwd_algo,
            &fwd_workspace_size);
        //printf("worksapce: %.2fMB\t", (float)fwd_workspace_size/1024/1024);
        if (retStatus != HIPDNN_STATUS_SUCCESS || (float)fwd_workspace_size/1024/1024 > 30000)
        {
          //printf("\n");
          continue;
        }
        ErrChk(hipMalloc((void **)&extra, fwd_workspace_size));

        float time_sum = 0.0;
        int count = 0;
        for (int i = 0; i < 1; i++)
        {
          hipEvent_t start, stop;
          hipEventCreate(&start);
          hipEventCreate(&stop) ;
          hipEventRecord(start, 0) ;
          ErrChk(hipdnnConvolutionForward(handle, &one, xDesc, x, filterDesc, filter, convDesc, fwd_algo, extra, fwd_workspace_size, &zero, yDesc, y));
          hipEventRecord(stop, 0) ;
          hipEventSynchronize(stop);
          float elapsedTime;
          hipEventElapsedTime(&elapsedTime, start, stop);
          if (i >= 0)
          {
            count++;
            time_sum += elapsedTime;
          }
          // printf("algo time cost: %f\n", double(end - start)/CLOCKS_PER_SEC);
        }
        
        //printf("algo time avg cost: %lf ms\n", double(time_sum)/count);
        
        hipFree(extra);
      }
      catch(const char *msg)
      {
        printf("%s", msg);
      }
      
    }

    hipFree(y);
    hipFree(filter);
    hipFree(x);
    free(h_Var0);
    free(h_Var1);
    free(h_Var2);
    ErrChk(hipdnnDestroyTensorDescriptor(xDesc));
    ErrChk(hipdnnDestroyTensorDescriptor(yDesc));
    ErrChk(hipdnnDestroyFilterDescriptor(filterDesc));
    ErrChk(hipdnnDestroyConvolutionDescriptor(convDesc));
  }
  end = clock();
  cout<<"time = "<<double(end-start)/CLOCKS_PER_SEC<<"s"<<endl;
  exit(EXIT_SUCCESS);
}

